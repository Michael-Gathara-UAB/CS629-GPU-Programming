#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// include kernels and cuda headers after definitions of structures
#include "kernels.cuh" 


void checkCUDAError(const char*);
void readRecords(student_record *records);
void studentRecordAOS2SOA(student_record *aos, student_records *soa);
void maximumMark_atomic(student_records*, student_records*, student_records*, student_records*);
void maximumMark_recursive(student_records*, student_records*, student_records*, student_records*);
void maximumMark_SM(student_records*, student_records*, student_records*, student_records*);
void maximumMark_shuffle(student_records*, student_records*, student_records*, student_records*);
void checkMaxCPU(student_records*);

#define THREADS_PER_BLOCK 256
#define NUM_BLOCKS (NUM_RECORDS+THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK

int main(void) {
	student_record *recordsAOS;
	student_records *h_records;
	student_records *h_records_result;
	student_records *d_records;
	student_records *d_records_result;
	
	//host allocation
	recordsAOS = (student_record*)malloc(sizeof(student_record)*NUM_RECORDS);
	h_records = (student_records*)malloc(sizeof(student_records));
	h_records_result = (student_records*)malloc(sizeof(student_records));

	//device allocation
	hipMalloc((void**)&d_records, sizeof(student_records));
	hipMalloc((void**)&d_records_result, sizeof(student_records));
	checkCUDAError("CUDA malloc");

	//read file
	readRecords(recordsAOS);
	studentRecordAOS2SOA(recordsAOS, h_records);
	
	//free AOS as it is no longer needed
	free(recordsAOS);

	//apply each approach in turn 
	maximumMark_atomic(h_records, h_records_result, d_records, d_records_result);
	maximumMark_recursive(h_records, h_records_result, d_records, d_records_result);
	maximumMark_SM(h_records, h_records_result, d_records, d_records_result);
	maximumMark_shuffle(h_records, h_records_result, d_records, d_records_result);


	// Cleanup
	free(h_records);
	free(h_records_result);
	hipFree(d_records);
	hipFree(d_records_result);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void readRecords(student_record *records){
	FILE *f = NULL;
	f = fopen("Student.dat", "rb"); //read and binary flags
	if (f == NULL){
		fprintf(stderr, "Error: Could not find file \n");
		exit(1);
	}

	//read student data
	if (fread(records, sizeof(student_record), NUM_RECORDS, f) != NUM_RECORDS){
		fprintf(stderr, "Error: Unexpected end of file!\n");
		exit(1);
	}
	fclose(f);
}

void studentRecordAOS2SOA(student_record *aos, student_records *soa){
	// Task 0.1 Convert student_record to student_records
    for(int i = 0; i<NUM_RECORDS; i++){
        soa->student_ids[i] = aos[i].student_id;
        soa->assignment_marks[i] = aos[i].assignment_mark;
    }
}

void checkMaxCPU(student_records *h_records){
    float max_cpu = 0;
    int max_cpu_id = 0;
    for (int i =0;i<NUM_RECORDS;i++){
        if(h_records->assignment_marks[i]>=max_cpu){
            max_cpu = h_records->assignment_marks[i];
            max_cpu_id = h_records->student_ids[i];
        }
    }
    printf("CPU : Highest mark recorded %f was by student %d\n", max_cpu, max_cpu_id);
}


void maximumMark_atomic(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;
	
	max_mark = 0;
	max_mark_student_id = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("Atomics: CUDA memcpy");

	hipEventRecord(start, 0);

	// Task 1.2 Confgure the kernel
    
   // int threadsPerBlock = 256;
   // int numBlocks = (NUM_RECORDS + threadsPerBlock -1)/threadsPerBlock;
   // defined outside 
   
	// Task 1.3) Launch and synchronize the kernel
    
    maximumMark_atomic_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>>(d_records);
    
	hipDeviceSynchronize();
    
	// Task 1.4)  Copy result back to host
	
    hipMemcpyFromSymbol(&max_mark,HIP_SYMBOL(d_max_mark),sizeof(float),0,hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&max_mark_student_id,HIP_SYMBOL(d_max_mark_student_id),sizeof(float),0,hipMemcpyDeviceToHost);
    
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// Task 1.5) Use CPU to validate results
    checkMaxCPU(h_records);
    
	//output result
	printf("Atomics: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//Task 2)
void maximumMark_recursive(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	int i;
	float max_mark;
	int max_mark_student_id;
	student_records *d_records_temp;
	float time;
	hipEvent_t start, stop;
	
	max_mark = 0;
	max_mark_student_id = 0.0f;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("Recursive: CUDA memcpy");

	hipEventRecord(start, 0);
	
    
	//Task 2.3) Recursively call GPU steps until there are THREADS_PER_BLOCK values left
    
   int records = NUM_RECORDS;
    
    while (records>=THREADS_PER_BLOCK){
        int num_blocks = (records + THREADS_PER_BLOCK -1)/THREADS_PER_BLOCK;
        maximumMark_recursive_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(d_records, d_records_result);
        hipDeviceSynchronize();
        d_records_temp = d_records;
        d_records = d_records_result;
        d_records_result = d_records_temp;
        records = records/2;
    }
    
    //Task 2.4) copy back the final THREADS_PER_BLOCK values
    hipMemcpy(h_records_result, d_records, sizeof(student_records), hipMemcpyDeviceToHost);
    
	//Task 2.5) reduce the final THREADS_PER_BLOCK values on CPU
    for (i = 0; i< records; i++){
        if(h_records_result->assignment_marks[i] >= max_mark){
            max_mark = h_records_result->assignment_marks[i];
            max_mark_student_id = h_records_result->student_ids[i];
        }
    }

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output the result
	printf("Recursive: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//Task 3)
void maximumMark_SM(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	unsigned int i;
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;
	
	max_mark = 0;
	max_mark_student_id = 0.0f;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("SM: CUDA memcpy");

	hipEventRecord(start, 0);
	
	//Task 3.4) Call the shared memory reduction kernel
    
    maximumMark_SM_kernel<<<NUM_BLOCKS,THREADS_PER_BLOCK>>>(d_records, d_records_result);
    hipDeviceSynchronize();
    
	//Task 3.5) Copy the final block values back to CPU
    
    hipMemcpy(h_records_result, d_records_result,  sizeof(student_records), hipMemcpyDeviceToHost);
    
	//Task 3.6) Reduce the block level results on CPU
    
    for (i = 0; i< NUM_RECORDS/2; i++){
        if(h_records_result->assignment_marks[i] >= max_mark){
            max_mark = h_records_result->assignment_marks[i];
            max_mark_student_id = h_records_result->student_ids[i];
        }
    }
    
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output result
	printf("SM: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//Task 4)
void maximumMark_shuffle(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	unsigned int i;
	unsigned int warps_per_grid;
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;
	
	max_mark = 0;
	max_mark_student_id = 0.0f;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("Shuffle: CUDA memcpy");
	
	hipEventRecord(start, 0);

	//Task 4.2) Execute the kernel, copy back result, reduce final values on CPU
    
    warps_per_grid = NUM_BLOCKS;
    maximumMark_shuffle_kernel<<<warps_per_grid,THREADS_PER_BLOCK>>>(d_records, d_records_result);
    hipDeviceSynchronize();
    
    hipMemcpy(h_records_result,d_records_result,sizeof(student_records), hipMemcpyDeviceToHost);
    
    for (i = 0; i< NUM_RECORDS; i++){
        if(h_records_result->assignment_marks[i] >= max_mark){
            max_mark = h_records_result->assignment_marks[i];
            max_mark_student_id = h_records_result->student_ids[i];
        }
    }
    
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output result
	printf("Shuffle: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}
