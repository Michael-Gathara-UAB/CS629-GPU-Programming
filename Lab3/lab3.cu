#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// include kernels and cuda headers after definitions of structures
#include "kernels.cuh" 


void checkCUDAError(const char*);
void readRecords(student_record *records);
void studentRecordAOS2SOA(student_record *aos, student_records *soa);
void maximumMark_atomic(student_records*, student_records*, student_records*, student_records*);
void maximumMark_recursive(student_records*, student_records*, student_records*, student_records*);
void maximumMark_SM(student_records*, student_records*, student_records*, student_records*);
void maximumMark_shuffle(student_records*, student_records*, student_records*, student_records*);


int main(void) {
	student_record *recordsAOS;
	student_records *h_records;
	student_records *h_records_result;
	student_records *d_records;
	student_records *d_records_result;
	
	//host allocation
	recordsAOS = (student_record*)malloc(sizeof(student_record)*NUM_RECORDS);
	h_records = (student_records*)malloc(sizeof(student_records));
	h_records_result = (student_records*)malloc(sizeof(student_records));

	//device allocation
	hipMalloc((void**)&d_records, sizeof(student_records));
	hipMalloc((void**)&d_records_result, sizeof(student_records));
	checkCUDAError("CUDA malloc");

	//read file
	readRecords(recordsAOS);
	studentRecordAOS2SOA(recordsAOS, h_records);
	
	//free AOS as it is no longer needed
	free(recordsAOS);

	//apply each approach in turn 
	maximumMark_atomic(h_records, h_records_result, d_records, d_records_result);
	// maximumMark_recursive(h_records, h_records_result, d_records, d_records_result);
	// maximumMark_SM(h_records, h_records_result, d_records, d_records_result);
	// maximumMark_shuffle(h_records, h_records_result, d_records, d_records_result);


	// Cleanup
	free(h_records);
	free(h_records_result);
	hipFree(d_records);
	hipFree(d_records_result);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void readRecords(student_record *records){
	FILE *f = NULL;
	f = fopen("Student.dat", "rb"); //read and binary flags
	if (f == NULL){
		fprintf(stderr, "Error: Could not find file \n");
		exit(1);
	}

	//read student data
	if (fread(records, sizeof(student_record), NUM_RECORDS, f) != NUM_RECORDS){
		fprintf(stderr, "Error: Unexpected end of file!\n");
		exit(1);
	}
	fclose(f);
}

void studentRecordAOS2SOA(student_record *aos, student_records *soa){
	for (int i = 0; i < NUM_RECORDS; i++) {
		soa->student_ids[i] = aos[i].student_id;
		soa->assignment_marks[i] = aos[i].assignment_mark;
	}
}

void maximumMark_atomic(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;
	
	max_mark = 0;
	max_mark_student_id = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("Atomics: CUDA memcpy");

	hipEventRecord(start, 0);

	// Task 1.2 Confgure the kernel
	int threadsPerBlock = 256;
	int blocksPerGrid = (NUM_RECORDS + threadsPerBlock - 1) / threadsPerBlock;

	// Task 1.3) Launch and synchronize the kernel
	hipDeviceSynchronize();
	maximumMark_atomic_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_records);
	hipDeviceSynchronize();
	
	// Task 1.4)  Copy result back to host
	hipMemcpyFromSymbol(&max_mark, HIP_SYMBOL(d_max_mark), sizeof(float), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&max_mark_student_id, HIP_SYMBOL(d_max_mark_student_id), sizeof(int), 0, hipMemcpyDeviceToHost);

	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	// Task 1.5) Use CPU to validate results

	float cpuMaxMark = 0.0f;
	int cpuMaxMarkStudentId = -1;
	for (int i = 0; i < NUM_RECORDS; i++) {
		if (h_records->assignment_marks[i] > cpuMaxMark) {
			cpuMaxMark = h_records->assignment_marks[i];
			cpuMaxMarkStudentId = h_records->student_ids[i];
		}
	}

	// printf("CPU: Highest mark recorded %f was by student %d\n", cpuMaxMark, cpuMaxMarkStudentId);

	// //output result
	// printf("Atomics: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	// printf("\tExecution time was %f ms\n", time);
	printf("%d", max_mark_student_id);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

__global__ void maximumMark_recursive_kernel(student_records *d_records, student_records *d_reduced_records, int num_records) {
    extern __shared__ student_record sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Task 2.1) Load student records into shared memory
    if (i < num_records) {
        sdata[tid].assignment_mark = d_records->assignment_marks[i];
        sdata[tid].student_id = d_records->student_ids[i];
        if (i + blockDim.x < num_records) {
            float nextMark = d_records->assignment_marks[i + blockDim.x];
            if (nextMark > sdata[tid].assignment_mark) {
                sdata[tid].assignment_mark = nextMark;
                sdata[tid].student_id = d_records->student_ids[i + blockDim.x];
            }
        }
    } else {
        sdata[tid].assignment_mark = -1.0;
        sdata[tid].student_id = -1;
    }
    __syncthreads();

    // Task 2.2) Compare two values and write the result to d_reduced_records
    // for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    //     if (tid < s && (i + s) < num_records) {
    //         if (sdata[tid + s].assignment_mark > sdata[tid].assignment_mark) {
    //             sdata[tid].assignment_mark = sdata[tid + s].assignment_mark;
    //             sdata[tid].student_id = sdata[tid + s].student_id;
    //         }
    //     }
    //     __syncthreads();
    // }

    // Write the result for this block to d_reduced_records
    if (tid == 0) {
        d_reduced_records->assignment_marks[blockIdx.x] = sdata[0].assignment_mark;
        d_reduced_records->student_ids[blockIdx.x] = sdata[0].student_id;
    }
}


//Task 3)
void maximumMark_SM(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	unsigned int i;
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;
	
	max_mark = 0;
	max_mark_student_id = 0.0f;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("SM: CUDA memcpy");

	hipEventRecord(start, 0);
	
	//Task 3.4) Call the shared memory reduction kernel
    //Task 3.5) Copy the final block values back to CPU
    hipMemcpy(h_records_result->assignment_marks, d_reduced_records, blocksPerGrid * sizeof(student_record), hipMemcpyDeviceToHost);

    //Task 3.6) Reduce the block level results on CPU
    for (i = 0; i < blocksPerGrid; i++) {
        if (h_records_result->assignment_marks[i] > max_mark) {
            max_mark = h_records_result->assignment_marks[i];
            max_mark_student_id = h_records_result->student_ids[i];
        }
    }

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output result
	printf("SM: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}

//Task 4)
void maximumMark_shuffle(student_records *h_records, student_records *h_records_result, student_records *d_records, student_records *d_records_result){
	unsigned int i;
	unsigned int warps_per_grid;
	float max_mark;
	int max_mark_student_id;
	float time;
	hipEvent_t start, stop;
	
	max_mark = 0;
	max_mark_student_id = 0.0f;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//memory copy records to device
	hipMemcpy(d_records, h_records, sizeof(student_records), hipMemcpyHostToDevice);
	checkCUDAError("Shuffle: CUDA memcpy");
	
	hipEventRecord(start, 0);

	//Task 4.2) Execute the kernel, copy back result, reduce final values on CPU

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	//output result
	printf("Shuffle: Highest mark recorded %f was by student %d\n", max_mark, max_mark_student_id);
	printf("\tExecution time was %f ms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}
