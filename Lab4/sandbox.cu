#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

#define N 4194304
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char*);
void random_ints(int *a);


// task 1.1
__device__ int d_a[N];
__device__ int d_b[N];
__device__ int d_c[N];

__global__ void vectorAdd(int max) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_c[i] = d_a[i] + d_b[i];
}

int main(void) {
    int *a, *b, *c;			// host copies of a, b, c
    int errors;
    unsigned int size = N * sizeof(int);

    // Alloc space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); random_ints(a);
    b = (int *)malloc(size); random_ints(b);
    c = (int *)malloc(size);

    // Copy inputs to device
    hipMemcpyToSymbol(HIP_SYMBOL(d_a), a, size);
    hipMemcpyToSymbol(HIP_SYMBOL(d_b), b, size);
    checkCUDAError("CUDA memcpy to symbol");

    // task 1.2 Record timings
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Launch add() kernel on GPU
    vectorAdd << <N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(N);
    checkCUDAError("CUDA kernel");

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    // Copy result back to host
    hipMemcpyFromSymbol(c,HIP_SYMBOL(d_c),size);
    checkCUDAError("CUDA memcpy");

    printf("Kernel Execution Time: %fms\n", ms);

    // task 1.3
    hipDeviceProp_t d_prop;
    int deviceId;
    double memoryClockRate, memoryBusWidth, theoreticalBW;

    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&d_prop,deviceId);

    memoryClockRate = d_prop.memoryClockRate * 1e-6;
    memoryBusWidth = d_prop.memoryBusWidth;
    theoreticalBW = ((memoryClockRate * memoryBusWidth) * 2)/8;

    printf("Theoretical Memory Bandwidth: %.2fGB/s\n",theoreticalBW);

    // task 1.4
    double r_bytes, w_bytes, measuredBW;
    r_bytes = N * 8;
    w_bytes = N * 4;
    measuredBW = ((r_bytes + w_bytes)/(ms/1000))/1e9 ;

    printf("Measured Memory Bandwidth: %.2fGB/s\n",measuredBW);

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(a); free(b); free(c);
    checkCUDAError("CUDA cleanup");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err)
    {
        fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void random_ints(int *a)
{
    for (unsigned int i = 0; i < N; i++){
        a[i] = rand();
    }
}