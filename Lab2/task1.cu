/*
Author: Michael Gathara (mikegtr at uab dot edu)*/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define BLOCK_SIZE 16  
void random_matrix(int *arr, int n);

__global__ void matrixMultGPUKernel(float *a, float *b, float *c, int m, int k, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < m && col < n) {
        float sum = 0.0;
        for (int i = 0; i < k; ++i) {
            sum += a[row * k + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

void matrixMultCPU(float *a, float *b, float *c, int m, int k, int n) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            float sum = 0.0;
            for (int x = 0; x < k; ++x) {
                sum += a[i * k + x] * b[x * n + j];
            }
            c[i * n + j] = sum;
        }
    }
}

void random_matrix(float *arr, int n) {
    for (unsigned int i = 0; i < n * n; i++) {
        arr[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int validate(float *c_gpu, float *c_cpu, int m, int n) {
    int errors = 0;
    for (int i = 0; i < m * n; i++) {
        // https://www.oreilly.com/library/view/c-in-a/0596006977/re57.html#:~:text=The%20fabs()%20function%20returns,%2C%20the%20function%20returns%20%2Dx%20.
        // Used the above link for fabs
        if (fabs(c_gpu[i] - c_cpu[i]) > 1) {
            // There might be soem floating point errors so I'm only checking to make sure the whole number section is the same
            errors++;
            printf("Mismatch at %d: GPU = %f, CPU = %f\n", i, c_gpu[i], c_cpu[i]);
        }
    }
    return errors;
}

int main() {
    int lim = 10000;
    unsigned int m = lim, k = lim, n = lim; 
    size_t a_size = m * k * sizeof(float);
    size_t b_size = k * n * sizeof(float);
    size_t c_size = m * n * sizeof(float);

    float *a, *b, *c_gpu, *c_cpu;
    a = (float*)malloc(a_size);
    b = (float*)malloc(b_size);
    c_gpu = (float*)malloc(c_size);
    c_cpu = (float*)malloc(c_size);

    random_matrix(a, m);
    random_matrix(b, k);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, a_size);
    hipMalloc(&d_b, b_size);
    hipMalloc(&d_c, c_size);

    hipMemcpy(d_a, a, a_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, b_size, hipMemcpyHostToDevice);

    int blockSizeArray[] = {8, 16, 32};
    for (int blockSizeIndex = 0; blockSizeIndex < 3; blockSizeIndex++) {
        int blockSize = blockSizeArray[blockSizeIndex];
        dim3 threads(blockSize, blockSize);
        dim3 grid((n + blockSize - 1) / blockSize, (m + blockSize - 1) / blockSize);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        matrixMultGPUKernel<<<grid, threads>>>(d_a, d_b, d_c, m, k, n);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        hipMemcpy(c_gpu, d_c, c_size, hipMemcpyDeviceToHost);
        matrixMultCPU(a, b, c_cpu, m, k, n);

        int errors = validate(c_gpu, c_cpu, m, n);
        printf("Block size %dx%d, Errors: %d\n", blockSize, blockSize, errors);

        float gflops = (m * k * n * 2.0) / 1e9 / (milliseconds / 1000);
        printf("Block size %dx%d, GFLOPS: %f\n", blockSize, blockSize, gflops);

        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    free(a); free(b); free(c_gpu); free(c_cpu);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
