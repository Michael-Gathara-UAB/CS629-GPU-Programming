/*
Author: Michael Gathara (mikegtr at uab dot edu)*/

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define BLOCK_SIZE 16  
void random_matrix(int *arr, int n);

__global__ void matrixMultGPUKernel(float *a, float *b, float *c, int m, int k, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < m && col < n) {
        float sum = 0.0;
        for (int i = 0; i < k; ++i) {
            sum += a[row * k + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

void matrixMultCPU(float *a, float *b, float *c, int m, int k, int n) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            float sum = 0.0;
            for (int x = 0; x < k; ++x) {
                sum += a[i * k + x] * b[x * n + j];
            }
            c[i * n + j] = sum;
        }
    }
}

void random_matrix(float *arr, int n) {
    for (unsigned int i = 0; i < n * n; i++) {
        arr[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int validate(float *c_gpu, float *c_cpu, int m, int n) {
    int errors = 0;
    for (int i = 0; i < m * n; i++) {
        // https://www.oreilly.com/library/view/c-in-a/0596006977/re57.html#:~:text=The%20fabs()%20function%20returns,%2C%20the%20function%20returns%20%2Dx%20.
        // Used the above link for fabs
        if (fabs(c_gpu[i] - c_cpu[i]) > 1e-6) {
            errors++;
            printf("Mismatch at %d: GPU = %f, CPU = %f\n", i, c_gpu[i], c_cpu[i]);
        }
    }
    return errors;
}

int main() {
    int lim = 32;
    unsigned int m = lim, k = lim, n = lim; 
    size_t a_size = m * k * sizeof(float);
    size_t b_size = k * n * sizeof(float);
    size_t c_size = m * n * sizeof(float);

    float *a, *b, *c_gpu, *c_cpu;
    a = (float*)malloc(a_size);
    b = (float*)malloc(b_size);
    c_gpu = (float*)malloc(c_size);
    c_cpu = (float*)malloc(c_size);

    random_matrix(a, m);
    random_matrix(b, k);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, a_size);
    hipMalloc(&d_b, b_size);
    hipMalloc(&d_c, c_size);

    hipMemcpy(d_a, a, a_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, b_size, hipMemcpyHostToDevice);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (m + BLOCK_SIZE - 1) / BLOCK_SIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrixMultGPUKernel<<<grid, threads>>>(d_a, d_b, d_c, m, k, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(c_gpu, d_c, c_size, hipMemcpyDeviceToHost);
    matrixMultCPU(a, b, c_cpu, m, k, n);

    int errors = validate(c_gpu, c_cpu, m, n);
		printf("There were %d errors", errors);

    float gflops = (m * k * n * 2) / 1e9  / (milliseconds / 1000);
    printf("\nGFLOPS: %f\n", gflops);

    free(a); free(b); free(c_gpu); free(c_cpu);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    hipEventDestroy(start); hipEventDestroy(stop);

    return 0;
}
