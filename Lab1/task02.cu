#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"


#define N 2050
#define THREADS_PER_BLOCK 128

void checkCUDAError(const char*);
void random_ints(int *a);
void vectorAddCPU(int *a, int *b, int *c);
int validate(int *c, int *c_ref);


__global__ 
void vectorAdd(int *a, int *b, int *c, int max) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < max) {	
		c[i] = a[i] + b[i];
	}
}

void vectorAddCPU(int *a, int *b, int *c_ref) {
	for (int i = 0; i < N; i++) {
		c_ref[i] = a[i] + b[i];
	}
}

int validate(int *c, int *c_ref) {
	int e = 0;
	for (int i = 0; i < N; i++) {
		if (c[i] != c_ref[i]) {
			printf("Error at %d: GOT {%d} in GPU and GOT {%d} in CPU", i, c[i], c_ref[i]);
			e++;
		}
	}
	return e;
}

int main(void) {
	int *a, *b, *c, *c_ref;			// host copies of a, b, c
	int *d_a, *d_b, *d_c;			// device copies of a, b, c
	int errors;
	unsigned int size = N * sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
	checkCUDAError("CUDA malloc");

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a);
	b = (int *)malloc(size); random_ints(b);
	c = (int *)malloc(size);
	c_ref = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy");

	// Launch add() kernel on GPU
	vectorAdd <<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(d_a, d_b, d_c, N);
	
	/* wait for all threads to complete */
	hipDeviceSynchronize();
	checkCUDAError("CUDA kernel");


	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy");
	
	vectorAddCPU(a, b, c_ref);
	errors = validate(c, c_ref);
	printf("Errors: %d\n", errors);	

	// Cleanup
	free(a); free(b); free(c); free(c_ref);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	checkCUDAError("CUDA cleanup");

	return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void random_ints(int *a)
{
	for (unsigned int i = 0; i < N; i++){
		a[i] = rand();
	}
}
