#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector_functions.h>
#include <hip/hip_vector_types.h>

#include "hip/hip_runtime.h"


#define HISTO_SIZE 256

void checkCUDAError(const char* msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void generate_data(char* data, unsigned int length) {
    for (int i = 0; i < length; ++i) {
        data[i] = (char)rand() % HISTO_SIZE;
    }
}

// took this from the slides, week 11 slide 53
__global__ void GPU_histogram(char* data, unsigned int length,
                              unsigned int* histo) {
    // Task 1.1 Add your implementation here
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < length) {
        int alph = data[i] - 'a';
        if (alph >= 0 && alph < 26) {
            atomicAdd(&histo[blockIdx.x * HISTO_SIZE + alph / 4], 1);
        }
        // if (alph >= 0 && alph < 26) {
        //     atomicAdd(&histo[alph / 4], 1);
        // }
    
    }
    if (blockIdx.x > 0) {
        __syncthreads();
        for (unsigned int bin = threadIdx.x; bin < HISTO_SIZE; bin += blockDim.x) {
            unsigned binVal = histo[blockIdx.x * HISTO_SIZE + bin];
            if (binVal > 0) {
                atomicAdd(&histo[bin], binVal);
            }
        }
    }
}

// took this from the slides, week 11 slide 38
void CPU_histogram(char* data, unsigned int length, unsigned int* histo) {
    // Task 1.2 Add a CPU implementation for verification
    for (unsigned int i = 0; i < length; ++i) {
        int alph = data[i] - 'a';
        if (alph >= 0 && alph < 26) {
            histo[alph / 4]++;
        } 

    }
}

/* Host code */
int main(void) {
    unsigned int input_length = 2048;
    char *h_data, *d_data;
    char* c_data;
    unsigned int* c_histo;
    unsigned int *h_histo, *d_histo;
    hipEvent_t start, stop;
    float ms;

    unsigned int data_size = input_length * sizeof(char);
    unsigned int histo_size = HISTO_SIZE * sizeof(unsigned int);

    // create timers
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate memory on the GPU for the output image
    hipMalloc((void**)&d_data, data_size);
    hipMalloc((void**)&d_histo, histo_size);
    hipMemset(d_histo, 0, histo_size);
    checkCUDAError("CUDA malloc");

    // allocate host data
    h_data = (char*)malloc(data_size);
    h_histo = (unsigned int*)malloc(histo_size);
    generate_data(h_data, input_length);

    // copy image to device memory
    hipMemcpy(d_data, h_data, input_length, hipMemcpyHostToDevice);
    checkCUDAError("CUDA memcpy to device");

    hipEventRecord(start, 0);
    // Task 1.3 Add kernel launch here...
    int threadsPerBlock = 256;
    int blocksPerGrid = (input_length + threadsPerBlock - 1) / threadsPerBlock;
    // GPU_histogram << <threadsPerBlock, (input_length + threadsPerBlock - 1) /
    // threadsPerBlock>> > (d_data, input_length, d_histo);
    GPU_histogram<<<blocksPerGrid, threadsPerBlock>>>(d_data, input_length,
                                                      d_histo);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    checkCUDAError("kernel normal");

    // copy the histogram back from the GPU
    hipMemcpy(h_histo, d_histo, histo_size, hipMemcpyDeviceToHost);
    checkCUDAError("CUDA memcpy from device");

    // Task 1.4 Verify output using a CPU function
    c_data = (char*)malloc(data_size);
    c_histo = (unsigned int*)malloc(histo_size);
    memcpy(c_data, h_data, data_size);
    // memcpy(c_histo, h_histo, histo_size);
    CPU_histogram(c_data, input_length, c_histo);

    for (int i = 0; i < HISTO_SIZE; ++i) {
        if (h_histo[i] != c_histo[i]) {
            printf("Error: GPU x CPU mismatch @ %d which is %d and %d\n", i,
                   h_histo[i], c_histo[i]);
            break;
        }
    }

	// THESE ARE FOR MY DEBUGGING
    // for (auto i = 0; i < input_length; ++i) {
    // 	printf("%d: %d\n", i, h_data[i]);
    // }

    // for (int i = 0; i < HISTO_SIZE; ++i) {
    //     printf("%d: %d\n", i, h_histo[i]);
    // }
    // printf("----------CPU----------\n");
    // for (int i = 0; i < HISTO_SIZE; ++i) {
    //     printf("%d: %d\n", i, c_histo[i]);
    // }

    // output timings
    printf("Execution time:");
    printf("\t%f\n", ms);

    // cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_data);
    hipFree(d_histo);
    free(h_data);
    free(h_histo);
    free(c_data);
    free(c_histo);

    return 0;
}