#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#define HISTO_SIZE 256


void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void generate_data(char * data, unsigned int length) {
	for (int i = 0; i < length; ++i) {
		data[i] = (char)rand() % HISTO_SIZE;
	}
}

__global__ void GPU_histogram(char * data, unsigned int length, unsigned int* histo) {
	// Task 1.1 Add your implementation here
}

void CPU_histogram(char * data, unsigned int length, unsigned int* histo) {
	// Task 1.2 Add a CPU implementation for verification

}

/* Host code */
int main(void) {
	unsigned int input_length = 2048;
	char * h_data, * d_data;
	unsigned int * h_histo, * d_histo;
	hipEvent_t start, stop;
	float ms;

	unsigned int data_size = input_length * sizeof(char);
	unsigned int histo_size = HISTO_SIZE * sizeof(unsigned int);

	// create timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate memory on the GPU for the output image
	hipMalloc((void**)&d_data, data_size);
	hipMalloc((void**)&d_histo, histo_size);
	hipMemset(d_histo, 0, histo_size);
	checkCUDAError("CUDA malloc");

	// allocate host data
	h_data = (char*)malloc(data_size);
	h_histo = (unsigned int*)malloc(histo_size);
	generate_data(h_data, input_length);

	// copy image to device memory
	hipMemcpy(d_data, h_data, input_length, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy to device");
	
	hipEventRecord(start, 0);
	// Task 1.3 Add kernel launch here...

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);
	checkCUDAError("kernel normal");


	// copy the histogram back from the GPU
	hipMemcpy(h_histo, d_histo, histo_size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy from device");

	//output timings
	printf("Execution time:");
	printf("\t%f\n", ms);

	// Task 1.4 Verify output using a CPU function
	

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_data);
	hipFree(d_histo);
	free(h_data);
	free(h_histo);

	return 0;
}



