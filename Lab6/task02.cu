#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void generate_data(float * data, unsigned int length) {
	for (int i = 0; i < length; ++i) {
		data[i] = (float)rand() / RAND_MAX;
	}
}

__global__ void GPU_scan(float * X, float * Y, unsigned int length) {
	// Task 2.1 Add your implementation here
}

void CPU_scan(float * X, float * Y, unsigned int length) {
	// Task 2.2 Add a CPU implementation for verification

}

/* Host code */
int main(void) {
	unsigned int input_length = 2048;
	float * h_input, * d_input, * h_output, * d_output;
	hipEvent_t start, stop;
	float ms;

	unsigned int data_size = input_length * sizeof(float);

	// create timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate memory on the GPU for the output image
	hipMalloc((void**)&d_input, data_size);
	hipMalloc((void**)&d_output, data_size);
	checkCUDAError("CUDA malloc");

	// allocate host data
	h_input = (float*)malloc(data_size);
	h_output = (float*)malloc(data_size);
	generate_data(h_input, input_length);

	// copy image to device memory
	hipMemcpy(d_input, h_input, input_length, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy to device");
	
	hipEventRecord(start, 0);
	// Task 2.3 Add kernel launch here...

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);
	checkCUDAError("kernel normal");


	// copy the histogram back from the GPU
	hipMemcpy(h_output, d_output, data_size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy from device");

	//output timings
	printf("Execution time:");
	printf("\t%f\n", ms);

	// Task 2.4 Verify output using a CPU function
	

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_input);
	hipFree(d_output);
	free(h_input);
	free(h_output);

	return 0;
}



