#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>


#define IMAGE_DIM 2048
constexpr int radius_one = 1;
constexpr int radius_two = 2;
constexpr int radius_three = 3;

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF     2e10f

using uchar = unsigned char; // from task1

void output_image_file(uchar3* image);
void input_image_file(char* filename, uchar3* image);
void checkCUDAError(const char *msg);


__global__ void image_blur_A(uchar3 *image, uchar3 *image_output) {
	// Add your implementation here
	extern __shared__ uchar3 shared_image[];

    int local_x = threadIdx.x + radius_one;
    int local_y = threadIdx.y + radius_one;
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global_idx = global_x + global_y * IMAGE_DIM;
    int local_idx = local_x + local_y * (blockDim.x + 2 * radius_one);

    shared_image[local_idx] = image[global_idx];

    if (threadIdx.x < radius_one) {
        int halo_idx = (global_x - radius_one + IMAGE_DIM) % IMAGE_DIM + global_y * IMAGE_DIM;
        shared_image[local_idx - radius_one] = image[halo_idx];
    }
    
	if (threadIdx.x >= blockDim.x - radius_one) {
        int halo_idx = (global_x + radius_one) % IMAGE_DIM + global_y * IMAGE_DIM;
        shared_image[local_idx + radius_one] = image[halo_idx];
    }
    
	if (threadIdx.y < radius_one) {
        int halo_idx = global_x + ((global_y - radius_one + IMAGE_DIM) % IMAGE_DIM) * IMAGE_DIM;
        shared_image[local_idx - radius_one * (blockDim.x + 2 * radius_one)] = image[halo_idx];
    }

    if (threadIdx.y >= blockDim.y - radius_one) {
        int halo_idx = global_x + ((global_y + radius_one) % IMAGE_DIM) * IMAGE_DIM;
        shared_image[local_idx + radius_one * (blockDim.x + 2 * radius_one)] = image[halo_idx];
    }

    __syncthreads();

    float3 sum = make_float3(0.0f, 0.0f, 0.0f);
    float factor = 1.0f / ((radius_one * 2 + 1) * (radius_one * 2 + 1));

    for (int dy = -radius_one; dy <= radius_one; dy++) {
        for (int dx = -radius_one; dx <= radius_one; dx++) {
            int idx = (local_x + dx) + (local_y + dy) * (blockDim.x + 2 * radius_one);
            uchar3 pixel = shared_image[idx];
            sum.x += pixel.x;
            sum.y += pixel.y;
            sum.z += pixel.z;
        }
    }

    sum.x *= factor;
    sum.y *= factor;
    sum.z *= factor;

    if (local_x < blockDim.x && local_y < blockDim.y) {  
        uchar3 output_pixel;
        output_pixel.x = static_cast<uchar>(sum.x);
        output_pixel.y = static_cast<uchar>(sum.y);
        output_pixel.z = static_cast<uchar>(sum.z);
        image_output[global_idx] = output_pixel;
    }
}

__global__ void image_blur_B(uchar3 *image, uchar3 *image_output) {
	// Add your implementation here
	extern __shared__ uchar3 shared_image[];

    int local_x = threadIdx.x + radius_two;
    int local_y = threadIdx.y + radius_two;
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global_idx = global_x + global_y * IMAGE_DIM;
    int local_idx = local_x + local_y * (blockDim.x + 2 * radius_two);

    shared_image[local_idx] = image[global_idx];

    if (threadIdx.x < radius_two) {
        int halo_idx = (global_x - radius_two + IMAGE_DIM) % IMAGE_DIM + global_y * IMAGE_DIM;
        shared_image[local_idx - radius_two] = image[halo_idx];
    }
    
	if (threadIdx.x >= blockDim.x - radius_two) {
        int halo_idx = (global_x + radius_two) % IMAGE_DIM + global_y * IMAGE_DIM;
        shared_image[local_idx + radius_two] = image[halo_idx];
    }
    
	if (threadIdx.y < radius_two) {
        int halo_idx = global_x + ((global_y - radius_two + IMAGE_DIM) % IMAGE_DIM) * IMAGE_DIM;
        shared_image[local_idx - radius_two * (blockDim.x + 2 * radius_two)] = image[halo_idx];
    }

    if (threadIdx.y >= blockDim.y - radius_two) {
        int halo_idx = global_x + ((global_y + radius_two) % IMAGE_DIM) * IMAGE_DIM;
        shared_image[local_idx + radius_two * (blockDim.x + 2 * radius_two)] = image[halo_idx];
    }

    __syncthreads();

    float3 sum = make_float3(0.0f, 0.0f, 0.0f);
    float factor = 1.0f / ((radius_two * 2 + 1) * (radius_two * 2 + 1));

    for (int dy = -radius_two; dy <= radius_two; dy++) {
        for (int dx = -radius_two; dx <= radius_two; dx++) {
            int idx = (local_x + dx) + (local_y + dy) * (blockDim.x + 2 * radius_two);
            uchar3 pixel = shared_image[idx];
            sum.x += pixel.x;
            sum.y += pixel.y;
            sum.z += pixel.z;
        }
    }

    sum.x *= factor;
    sum.y *= factor;
    sum.z *= factor;

    if (local_x < blockDim.x && local_y < blockDim.y) {  
        uchar3 output_pixel;
        output_pixel.x = static_cast<uchar>(sum.x);
        output_pixel.y = static_cast<uchar>(sum.y);
        output_pixel.z = static_cast<uchar>(sum.z);
        image_output[global_idx] = output_pixel;
    }

	
}

__global__ void image_blur_C(uchar3 *image, uchar3 *image_output) {
	// Add your implementation here
	extern __shared__ uchar3 shared_image[];

    int local_x = threadIdx.x + radius_three;
    int local_y = threadIdx.y + radius_three;
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global_idx = global_x + global_y * IMAGE_DIM;
    int local_idx = local_x + local_y * (blockDim.x + 2 * radius_three);

    shared_image[local_idx] = image[global_idx];

    if (threadIdx.x < radius_three) {
        int halo_idx = (global_x - radius_three + IMAGE_DIM) % IMAGE_DIM + global_y * IMAGE_DIM;
        shared_image[local_idx - radius_three] = image[halo_idx];
    }
    
	if (threadIdx.x >= blockDim.x - radius_three) {
        int halo_idx = (global_x + radius_three) % IMAGE_DIM + global_y * IMAGE_DIM;
        shared_image[local_idx + radius_three] = image[halo_idx];
    }
    
	if (threadIdx.y < radius_three) {
        int halo_idx = global_x + ((global_y - radius_three + IMAGE_DIM) % IMAGE_DIM) * IMAGE_DIM;
        shared_image[local_idx - radius_three * (blockDim.x + 2 * radius_three)] = image[halo_idx];
    }

    if (threadIdx.y >= blockDim.y - radius_three) {
        int halo_idx = global_x + ((global_y + radius_three) % IMAGE_DIM) * IMAGE_DIM;
        shared_image[local_idx + radius_three * (blockDim.x + 2 * radius_three)] = image[halo_idx];
    }

    __syncthreads();

    float3 sum = make_float3(0.0f, 0.0f, 0.0f);
    float factor = 1.0f / ((radius_three * 2 + 1) * (radius_three * 2 + 1));

    for (int dy = -radius_three; dy <= radius_three; dy++) {
        for (int dx = -radius_three; dx <= radius_three; dx++) {
            int idx = (local_x + dx) + (local_y + dy) * (blockDim.x + 2 * radius_three);
            uchar3 pixel = shared_image[idx];
            sum.x += pixel.x;
            sum.y += pixel.y;
            sum.z += pixel.z;
        }
    }

    sum.x *= factor;
    sum.y *= factor;
    sum.z *= factor;

    if (local_x < blockDim.x && local_y < blockDim.y) {  
        uchar3 output_pixel;
        output_pixel.x = static_cast<uchar>(sum.x);
        output_pixel.y = static_cast<uchar>(sum.y);
        output_pixel.z = static_cast<uchar>(sum.z);
        image_output[global_idx] = output_pixel;
    }
}


/* Host code */

int main(void) {
	unsigned int image_size;
	uchar3 *d_image, *d_image_output;
	uchar3 *h_image;
	hipEvent_t start, stop;
	float ms;

	image_size = IMAGE_DIM*IMAGE_DIM*sizeof(uchar3);

	// create timers
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// allocate memory on the GPU for the output image
	hipMalloc((void**)&d_image, image_size);
	hipMalloc((void**)&d_image_output, image_size);
	checkCUDAError("CUDA malloc");

	// allocate and load host image
	h_image = (uchar3*)malloc(image_size);
	input_image_file("input.ppm", h_image);

	// copy image to device memory
	hipMemcpy(d_image, h_image, image_size, hipMemcpyHostToDevice);
	checkCUDAError("CUDA memcpy to device");

	//cuda layout and execution
	dim3    blocksPerGrid(IMAGE_DIM / 16, IMAGE_DIM / 16);
	dim3    threadsPerBlock(16, 16);

	// normal version
	hipEventRecord(start, 0);
	// Uncomment each line to test each kernel
	// image_blur_A << <blocksPerGrid, threadsPerBlock >> >(d_image, d_image_output);
	image_blur_A<<<blocksPerGrid, threadsPerBlock, (16 + 2 * radius_one) * (16 + 2 * radius_one) * sizeof(uchar3)>>>(d_image, d_image_output);

	// image_blur_B << <blocksPerGrid, threadsPerBlock >> >(d_image, d_image_output);
	image_blur_B<<<blocksPerGrid, threadsPerBlock, (16 + 2 * radius_two) * (16 + 2 * radius_two) * sizeof(uchar3)>>>(d_image, d_image_output);

	// image_blur_C << <blocksPerGrid, threadsPerBlock >> >(d_image, d_image_output);
	image_blur_C<<<blocksPerGrid, threadsPerBlock, (16 + 2 * radius_three) * (16 + 2 * radius_three) * sizeof(uchar3)>>>(d_image, d_image_output);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ms, start, stop);
	checkCUDAError("kernel normal");


	// copy the image back from the GPU for output to file
	hipMemcpy(h_image, d_image_output, image_size, hipMemcpyDeviceToHost);
	checkCUDAError("CUDA memcpy from device");

	//output timings
	printf("Execution time:");
	printf("\t%f\n", ms);

	// output image
	output_image_file(h_image);

	//cleanup
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_image);
	hipFree(d_image_output);
	free(h_image);

	return 0;
}

void output_image_file(uchar3* image)
{
	FILE *f; //output file handle

	//open the output file and write header info for PPM filetype
	f = fopen("output.ppm", "wb");
	if (f == NULL){
		fprintf(stderr, "Error opening 'output.ppm' output file\n");
		exit(1);
	}
	fprintf(f, "P6\n");
	fprintf(f, "# CS 629/729 Lab 05 Task02\n");
	fprintf(f, "%d %d\n%d\n", IMAGE_DIM, IMAGE_DIM, 255);
	for (int x = 0; x < IMAGE_DIM; x++){
		for (int y = 0; y < IMAGE_DIM; y++){
			int i = x + y*IMAGE_DIM;
			fwrite(&image[i], sizeof(unsigned char), 3, f);
		}
	}

	fclose(f);
}

void input_image_file(char* filename, uchar3* image)
{
	FILE *f; //input file handle
	char temp[256];
	unsigned int x, y, s;

	//open the input file and write header info for PPM filetype
	f = fopen("input.ppm", "rb");
	if (f == NULL){
		fprintf(stderr, "Error opening 'input.ppm' input file\n");
		exit(1);
	}
	fscanf(f, "%s\n", &temp);
	fscanf(f, "%d %d\n", &x, &y);
	fscanf(f, "%d\n",&s);
	if ((x != y) && (x != IMAGE_DIM)){
		fprintf(stderr, "Error: Input image file has wrong fixed dimensions\n");
		exit(1);
	}

	for (int x = 0; x < IMAGE_DIM; x++){
		for (int y = 0; y < IMAGE_DIM; y++){
			int i = x + y*IMAGE_DIM;
			fread(&image[i], sizeof(unsigned char), 3, f);
		}
	}

	fclose(f);
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
